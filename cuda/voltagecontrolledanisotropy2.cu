#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"
#include "amul.h"

// Add voltage-controlled magnetic anisotropy field to B.
// https://www.nature.com/articles/s42005-019-0189-6.pdf
extern "C" __global__ void
addvoltagecontrolledanisotropy2(float* __restrict__  Bx, float* __restrict__  By, float* __restrict__  Bz,
                       float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
                       float* __restrict__ Ms_, float Ms_mul,
                       float* __restrict__ vcmaCoeff_, float vcmaCoeff_mul,
                       float* __restrict__ voltage_, float voltage_mul,
                       float* __restrict__ ux_, float ux_mul,
                       float* __restrict__ uy_, float uy_mul,
                       float* __restrict__ uz_, float uz_mul,
                       int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 u   = normalized(vmul(ux_, uy_, uz_, ux_mul, uy_mul, uz_mul, i));
        float invMs = inv_Msat(Ms_, Ms_mul, i);
        float  vcmaCoeff  = amul(vcmaCoeff_, vcmaCoeff_mul, i) * invMs;
        float  voltage  = amul(voltage_, voltage_mul, i) * invMs;
        float3 m   = {mx[i], my[i], mz[i]};
        float  mu  = dot(m, u);
        float3 Ba  = 2.0f*vcmaCoeff*voltage*    (mu)*u;

        Bx[i] += Ba.x;
        By[i] += Ba.y;
        Bz[i] += Ba.z;
    }
}

